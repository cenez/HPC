#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define ERR 1e-6

void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        assert(fabs(out[i] - a[i] - b[i]) < ERR);
    }
    printf("OK\n");
}
__device__ void task(int n){
	long x = 0;
	for(int i = 0; i < n; i++)
		x = i+1;
	n = x;
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
	int idx = threadIdx.x;
	int bdim = blockDim.x; 
	//printf("id=%d, bdim=%d\n", idx, bdim);

    for(int i = idx; i < n; i += bdim){
        out[i] = a[i] + b[i];
		task(n);
    }
}

int main(int argc, char *argv[]){
	if(argc<2) return 0;
	long N = atol(argv[1]);
	printf("############### Teste de desempenho para GPU(%ld)\n", N);

    float *a, *b, *out; float *d_a, *d_b, *d_out;
    
	a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    
	for(int i = 0; i < N; i++){ a[i] = i; b[i] = i; }
	
	clock_t start = clock();

	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	
	vector_add<<<1,256>>>(d_out, d_a, d_b, N);
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

	clock_t end = clock();
	float seconds = (float)(end - start) / 10000000; // / CLOCKS_PER_SEC;
    printf("Time: %.5f seconds CUDA. Verify: ", seconds);

	verify(out, a, b, N);
	
	free(a);
    free(b);
    free(out);
}

